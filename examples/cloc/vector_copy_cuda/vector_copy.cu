#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

extern "C" __global__ void vector_copy(__device__ int *in, __device__ int *out) {
  int id = blockIdx.x; 
  out[id] = in[id];
}
