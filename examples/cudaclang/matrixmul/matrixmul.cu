#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//

#include <stdio.h>

#define N 9
#define M 9
#define P 9

__global__
void matrixMul(int *matrixA, int *matrixB, int *matrixC,
               int ARows, int ACols, int BCols )
{
  int i = blockIdx.x;
  int j = blockIdx.y;

  if (i < ARows && j < BCols)
  {
    int value = 0;
    for (int k = 0; k < ACols; ++k)
    {
      value += matrixA[i*ACols+k] * matrixB[k*BCols+j];
    }
    matrixC[i*BCols+j] = value;
  }
}

void printMatrix(int *matrix, int Rows, int Cols)
{
  for (int i = 0; i < Rows; ++i)
  {
    printf("\n[");
    bool first = true;
    for (int j = 0; j < Cols; ++j)
    {
      if (first)
      {
        printf("%d", matrix[i*Cols+j]);
        first = false;
      }
      else
      {
        printf(", %d", matrix[i*Cols+j]);
      }
    }
    printf("]");
  }
}

void printCudaError(hipError_t error)
{
  printf("Cuda Error: %s\n", hipGetErrorString(error));
}

void randomizeMatrix(int *matrix, int Rows, int Cols)
{
  for (int i = 0; i < Rows*Cols; ++i)
    matrix[i] = rand() % 10;
}

void clearMatrix(int *matrix, int Rows, int Cols )
{
  for (int i = 0; i < Rows*Cols; ++i)
    matrix[i] = 0;
}
bool cudaCallSuccessful(hipError_t error)
{
  if (error != hipSuccess)
    printCudaError(error);
  return error == hipSuccess;
}

bool deviceCanCompute(int deviceID)
{
  bool canCompute = false;
  hipDeviceProp_t deviceProp;
  bool devicePropIsAvailable =
    cudaCallSuccessful(hipGetDeviceProperties(&deviceProp, deviceID));
  if (devicePropIsAvailable)
  {
    canCompute = deviceProp.computeMode != hipComputeModeProhibited;
    if (!canCompute)
      printf("Compute mode is prohibited\n");
  }
  return canCompute;
}

bool deviceIsAvailable(int *deviceID)
{
  return cudaCallSuccessful(hipGetDevice(deviceID));
}

// We always use device 0
bool haveComputeDevice()
{
  int deviceID = 0;
  return deviceIsAvailable(&deviceID) && deviceCanCompute(deviceID);
}

int main()
{
  int hostSrcMatA[N*M];
  int hostSrcMatB[M*P];
  int hostDstMat[N*P];

  if (!haveComputeDevice())
  {
    printf("No compute device available\n");
    return 0;
  }

  randomizeMatrix(hostSrcMatA, N, M);
  randomizeMatrix(hostSrcMatB, M, P);
  clearMatrix(hostDstMat, N, P);

  printf("A: ");
  printMatrix(hostSrcMatA, N, M);
  printf("\nB: ");
  printMatrix(hostSrcMatB, M ,P);
  printf("\n");

  int *deviceSrcMatA = NULL;
  int *deviceSrcMatB = NULL;
  int *deviceDstMat = NULL;

  bool matrixAAllocated =
    cudaCallSuccessful(hipMalloc((void **)&deviceSrcMatA, N*M*sizeof(int)));
  bool matrixBAllocated =
    cudaCallSuccessful(hipMalloc((void **)&deviceSrcMatB, M*P*sizeof(int)));
  bool matrixCAllocated =
    cudaCallSuccessful(hipMalloc((void **)&deviceDstMat, N*P*sizeof(int)));

  if (matrixAAllocated && matrixBAllocated && matrixCAllocated)
  {
    bool copiedSrcMatA =
      cudaCallSuccessful(hipMemcpy(deviceSrcMatA, hostSrcMatA,
                                    N*M*sizeof(int),
                                    hipMemcpyHostToDevice));
    bool copiedSrcMatB =
      cudaCallSuccessful(hipMemcpy(deviceSrcMatB, hostSrcMatB,
                                    M*P*sizeof(int),
                                    hipMemcpyHostToDevice));

    if (copiedSrcMatA && copiedSrcMatB)
    {
        dim3 dimGrid(N,P);
        matrixMul<<<dimGrid, 1>>>(deviceSrcMatA, deviceSrcMatB, deviceDstMat,
                                  N, M, P);
      if (cudaCallSuccessful(hipMemcpy(hostDstMat,
                                        deviceDstMat,
                                        N*P*sizeof(int),
                                        hipMemcpyDeviceToHost)))
      {
        printf("Mul: ");
        printMatrix(hostDstMat, N, P);
        printf("\n");
      }
      else
      {
        printf("Unable to copy memory from device to host\n");
      }
    }
  }

  if (matrixAAllocated)
    hipFree(deviceSrcMatA);
  if (matrixBAllocated)
    hipFree(deviceSrcMatB);
  if (matrixCAllocated)
    hipFree(deviceDstMat);

  return 0;
}
