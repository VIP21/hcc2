#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//

#include <stdio.h>

#define N 10

__global__
void addVector(int *vectorA, int *vectorB, int*vectorC)
{
  int i = blockIdx.x;
  if (i<N) {
    vectorC[i] = vectorA[i] + vectorB[i];
  }
}

void printVector(int *vector)
{
  printf("[");
  bool first = true;
  for (int i = 0; i<N; ++i)
  {
    if (first)
    {
      printf("%d", vector[i]);
      first = false;
    }
    else
    {
      printf(", %d", vector[i]);
    }
  }
  printf("]");
}

void printCudaError(hipError_t error)
{
  printf("Cuda Error: %s\n", hipGetErrorString(error));
}

void randomizeVector(int *vector)
{
  for (int i = 0; i < N; ++i)
    vector[i] = rand() % 10;
}

void clearVector(int *vector)
{
  for (int i = 0; i < N; ++i)
    vector[i] = 0;
}
bool cudaCallSuccessful(hipError_t error)
{
  if (error != hipSuccess)
    printCudaError(error);
  return error == hipSuccess;
}

bool deviceCanCompute(int deviceID)
{
  bool canCompute = false;
  hipDeviceProp_t deviceProp;
  bool devicePropIsAvailable =
    cudaCallSuccessful(hipGetDeviceProperties(&deviceProp, deviceID));
  if (devicePropIsAvailable)
  {
    canCompute = deviceProp.computeMode != hipComputeModeProhibited;
    if (!canCompute)
      printf("Compute mode is prohibited\n");
  }
  return canCompute;
}

bool deviceIsAvailable(int *deviceID)
{
  return cudaCallSuccessful(hipGetDevice(deviceID));
}

// We always use device 0
bool haveComputeDevice()
{
  int deviceID = 0;
  return deviceIsAvailable(&deviceID) && deviceCanCompute(deviceID);
}

int main()
{
  int hostSrcVecA[N];
  int hostSrcVecB[N];
  int hostDstVec[N];

  if (!haveComputeDevice())
  {
    printf("No compute device available\n");
    return 0;
  }

  randomizeVector(hostSrcVecA);
  randomizeVector(hostSrcVecB);
  clearVector(hostDstVec);

  printf("  A: ");
  printVector(hostSrcVecA);
  printf("\n  B: ");
  printVector(hostSrcVecB);
  printf("\n");

  int *deviceSrcVecA = NULL;
  int *deviceSrcVecB = NULL;
  int *deviceDstVec = NULL;

  bool vectorAAllocated =
    cudaCallSuccessful(hipMalloc((void **)&deviceSrcVecA, N*sizeof(int)));
  bool vectorBAllocated =
    cudaCallSuccessful(hipMalloc((void **)&deviceSrcVecB, N*sizeof(int)));
  bool vectorCAllocated =
    cudaCallSuccessful(hipMalloc((void **)&deviceDstVec, N*sizeof(int)));

  if (vectorAAllocated && vectorBAllocated && vectorCAllocated)
  {
    bool copiedSrcVecA =
      cudaCallSuccessful(hipMemcpy(deviceSrcVecA, hostSrcVecA,
                                    N * sizeof(int), hipMemcpyHostToDevice));
    bool copiedSrcVecB =
      cudaCallSuccessful(hipMemcpy(deviceSrcVecB, hostSrcVecB,
                                    N * sizeof(int), hipMemcpyHostToDevice));

    if (copiedSrcVecA && copiedSrcVecB)
    {
      addVector<<<N, 1>>>(deviceSrcVecA, deviceSrcVecB, deviceDstVec);

      if (cudaCallSuccessful(hipMemcpy(hostDstVec,
                                        deviceDstVec,
                                        N * sizeof(int),
                                        hipMemcpyDeviceToHost)))
      {
        printf("Sum: ");
        printVector(hostDstVec);
        printf("\n");
      }
      else
      {
        printf("Unable to copy memory from device to host\n");
      }
    }
  }

  if (vectorAAllocated)
    hipFree(deviceSrcVecA);
  if (vectorBAllocated)
    hipFree(deviceSrcVecB);
  if (vectorCAllocated)
    hipFree(deviceDstVec);

  return 0;
}
