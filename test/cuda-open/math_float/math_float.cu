#include "hip/hip_runtime.h"
// MIT License
//
// Copyright (c) 2018 Advanced Micro Devices, Inc. All Rights Reserved.
//
// Permission is hereby granted, free of charge, to any person
// obtaining a copy of this software and associated documentation
// files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use, copy,
// modify, merge, publish, distribute, sublicense, and/or sell copies
// of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be
// included in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
// MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS
// BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN
// ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
// CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

// These test only check if the code compiles, we don't test
// functionality yet.
// Reference: Cuda Toolkit v 9.2.88
//  1.3 Single Presicion Mathematical Functions
//  1.5 Single Presicion Intrinsics
#include <stdio.h>
#include <hip/hip_host_runtime_api.h>
#define N 10

__global__
void testFloatMath(float *b)
{
  int i = blockIdx.x;
  float f = (float) i;
  float dummy;
  float dummy2;
  int idummy;
  if (i<N) {
    // 1.3 Single Presicion Mathematical Functions
    b[i] = acosf(f);
    b[i] += acoshf(f);
    b[i] += asinf(f);
    b[i] += asinhf(f);
    b[i] += atan2f(f,f);
    b[i] += atanf(f);
    b[i] += atanhf(f);
    b[i] += cbrtf(f);
    b[i] += ceilf(f);
    //b[i] += copysign(f, -f); // Fixme: Add to cuda_open headers
    b[i] += cosf(f);
    b[i] += coshf(f);
    b[i] += cospif(f);
    b[i] += cyl_bessel_i0f(f);
    b[i] += cyl_bessel_i1f(f);
    b[i] += erfcf(f);
    b[i] += erfcinvf(f);
    b[i] += erfcxf(f);
    b[i] += erff(f);
    b[i] += erfinvf(f);
    b[i] += exp10f(f);
    b[i] += exp2f(f);
    b[i] += expf(f);
    b[i] += expm1f(f);
    b[i] += fabsf(f);
    b[i] += fdimf(f,f);
    b[i] += fdividef(f,f);
    b[i] += floorf(f);
    b[i] += fmaf(f,f,f);
    b[i] += fmaxf(f,f);
    b[i] += fminf(f,f);
    b[i] += fmodf(f,f);
    // b[i] += frexpf(f, &idummy); // Fixme: missing function _nv_frexpf
    b[i] += hypotf(f,f);
    b[i] += (float) ilogbf(f); 
    // b[i] += isfinite(f); // Fixme: Add to cuda_open headers
    // b[i] += isinf(f); // Fixme: Add to cuda_open headers
    // b[i] += isnan(f); // Fixme: Add to cuda_open headers
    b[i] += j0f(f);
    b[i] += j1f(f);
    // b[i] += jnf(1,f); // Fixme: missing function _nv_jnf
    b[i] += ldexpf(f,1);
    b[i] += lgammaf(f);
    b[i] += (float) llrintf(f);
    b[i] += (float) llroundf(f);
    b[i] += log10f(f);
    b[i] += log1pf(f);
    b[i] += log2f(f);
    b[i] += logbf(f);
    b[i] += logf(f);
    b[i] += (float) lrintf(f);
    b[i] += (float) lroundf(f);
    // b[i] += modff(f, &dummy); // Fixme: missing function _nv_modff
    // b[i] += nanf(""); // Fixme: missing function _nv_jnf
    b[i] += nearbyintf(f);
    b[i] += nextafterf(f,f);
    b[i] += norm3df(f,f,f);
    b[i] += norm4df(f,f,f,f);
    b[i] += normcdff(f);
    b[i] += normcdfinvf(f);
    // b[i] += normf(1,f); // Fixme: const -> non const conversion
    b[i] += powf(f,f);
    b[i] += rcbrtf(f);
    b[i] += remainderf(f,f); 
    // b[i] += remquof(f,f, &idummy); // Fixme: missing function __nv_remquof
    b[i] += rhypotf(f,f);
    b[i] += rintf(f);
    // b[i] += rnorm3df(f,f,f); // Fixme: missing function __nv_rnorm3df
    // b[i] += rnorm4df(f,f,f,f); // Fixme: missing function __nv_rnorm4df
    // b[i] += rnormf(1, &f); // Fixme: missing function __nv_rnormf
    b[i] += roundf(f);
    b[i] += rsqrtf(f);
    //b[i] += scalblnf(f, 1); // Fixme: missing function __nv_scalbnf
    //b[i] += scalbnf(f, 1);  // Fixme: missing function __nv_scalbnf
    // b[i] += signbit(f); // Fixme: Add to cuda_open headers
    // sincosf(f, &dummy, &dummy2); // Fixme: missing function __nv_sincosf
    // sincospif(f, &dummy, &dummy2); // Fixme: missing function __nv_sincospif
    b[i] += sinf(f);
    b[i] += sinhf(f);
    b[i] += sinpif(f);
    b[i] += sqrtf(f);
    b[i] += tanf(f);
    b[i] += tanhf(f);
    b[i] += tgammaf(f);
    b[i] += truncf(f);
    b[i] += y0f(f);
    b[i] += y1f(f);
    // b[i] += ynf(1,f); // Fixme: missing function __nv_ynf

   // 1.5 Single Presicion Intrinsics
    // FIXME: Add instrinsics
  }
}

void printArray(float *array)
{
  printf("[");
  bool first = true;
  for (int i = 0; i<N; ++i)
  {
    if (first)
    {
      printf("%f", array[i]);
      first = false;
    }
    else
    {
      printf(", %f", array[i]);
    }
  }
  printf("]");
}

void printHipError(hipError_t error)
{
  printf("Hip Error: %s\n", hipGetErrorString(error));
}

bool hipCallSuccessful(hipError_t error)
{
  if (error != hipSuccess)
    printHipError(error);
  return error == hipSuccess;
}

bool deviceCanCompute(int deviceID)
{
  bool canCompute = false;
  hipDeviceProp_t deviceProp;
  bool devicePropIsAvailable =
    hipCallSuccessful(hipGetDeviceProperties(&deviceProp, deviceID));
  if (devicePropIsAvailable)
  {
    canCompute = deviceProp.computeMode != hipComputeModeProhibited;
    if (!canCompute)
      printf("Compute mode is prohibited\n");
  }
  return canCompute;
}

bool deviceIsAvailable(int *deviceID)
{
  return hipCallSuccessful(hipGetDevice(deviceID));
}

// We always use device 0
bool haveComputeDevice()
{
  int deviceID = 0;
  return deviceIsAvailable(&deviceID) && deviceCanCompute(deviceID);
}

int main()
{

  float hostArray[N];

  if (!haveComputeDevice())
  {
    printf("No compute device available\n");
    return 0;
  }

  for (int i = 0; i<N; ++i)
    hostArray[i] = 0.0;

  printf("Array content before kernel:\n");
  printArray(hostArray);
  printf("\n");

  float *deviceArray;
  if (!hipCallSuccessful(hipMalloc((void **)&deviceArray, N*sizeof(float))))
  {
    printf("Unable to allocate device memory\n");
    return 0;
  }

  hipLaunchKernelGGL((testFloatMath), dim3(N), dim3(1), 0, 0, deviceArray);

  if (hipCallSuccessful(hipMemcpy(hostArray,
                                     deviceArray,
                                     N * sizeof(float),
                                     hipMemcpyDeviceToHost)))
  {
    printf("Array content after kernel:\n");
    printArray(hostArray);
    printf("\n");
  }
  else
  {
    printf("Unable to copy memory from device to host\n");
  }

  hipFree(deviceArray);
  return 0;
}
