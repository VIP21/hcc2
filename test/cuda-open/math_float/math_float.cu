#include "hip/hip_runtime.h"
// MIT License
//
// Copyright (c) 2018 Advanced Micro Devices, Inc. All Rights Reserved.
//
// Permission is hereby granted, free of charge, to any person
// obtaining a copy of this software and associated documentation
// files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use, copy,
// modify, merge, publish, distribute, sublicense, and/or sell copies
// of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be
// included in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
// MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS
// BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN
// ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
// CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

// These test only check if the code compiles, we don't test
// functionality yet.
// Reference: Cuda Toolkit v 9.2.88
//  1.3 Single Presicion Mathematical Functions
//  1.5 Single Presicion Intrinsics
#include <stdio.h>
#include <hip/hip_host_runtime_api.h>
#define N 10

__global__
void testFloatMath(float *b)
{
  int i = blockIdx.x;
  float f = (float) i;
  float dummy;
  float dummy2;
  int idummy;
  if (i<N) {
    // 1.3 Single Presicion Mathematical Functions
    b[i] = acosf(f);
    b[i] += acoshf(f);
    b[i] += asinf(f);
    b[i] += asinhf(f);
    b[i] += atan2f(f,f);
    b[i] += atanf(f);
    b[i] += atanhf(f);
    b[i] += cbrtf(f);
    b[i] += ceilf(f);
    //b[i] += copysign(f, -f); // Fixme: Add to cuda_open headers
    b[i] += cosf(f);
    b[i] += coshf(f);
    b[i] += cospif(f);
    b[i] += cyl_bessel_i0f(f);
    b[i] += cyl_bessel_i1f(f);
    b[i] += erfcf(f);
    b[i] += erfcinvf(f);
    b[i] += erfcxf(f);
    b[i] += erff(f);
    b[i] += erfinvf(f);
    b[i] += exp10f(f);
    b[i] += exp2f(f);
    b[i] += expf(f);
    b[i] += expm1f(f);
    b[i] += fabsf(f);
    b[i] += fdimf(f,f);
    b[i] += fdividef(f,f);
    b[i] += floorf(f);
    b[i] += fmaf(f,f,f);
    b[i] += fmaxf(f,f);
    b[i] += fminf(f,f);
    b[i] += fmodf(f,f);
    // b[i] += frexpf(f, &idummy); // Fixme: missing function _nv_frexpf
    b[i] += hypotf(f,f);
    b[i] += (float) ilogbf(f);
    b[i] += isfinite(f);
    b[i] += isinf(f);
    b[i] += isnan(f);
    b[i] += j0f(f);
    b[i] += j1f(f);
    // b[i] += jnf(1,f); // Fixme: missing function _nv_jnf
    b[i] += ldexpf(f,1);
    b[i] += lgammaf(f);
    b[i] += (float) llrintf(f);
    b[i] += (float) llroundf(f);
    b[i] += log10f(f);
    b[i] += log1pf(f);
    b[i] += log2f(f);
    b[i] += logbf(f);
    b[i] += logf(f);
    b[i] += (float) lrintf(f);
    b[i] += (float) lroundf(f);
    // b[i] += modff(f, &dummy); // Fixme: missing function _nv_modff
    // b[i] += nanf(""); // Fixme: Add to cuda_open headers
    b[i] += nearbyintf(f);
    b[i] += nextafterf(f,f);
    b[i] += norm3df(f,f,f);
    b[i] += norm4df(f,f,f,f);
    b[i] += normcdff(f);
    b[i] += normcdfinvf(f);
    //    b[i] += normf(1,&f); // Fixme: missing function __nv_normf
    b[i] += powf(f,f);
    b[i] += rcbrtf(f);
    b[i] += remainderf(f,f);
    // b[i] += remquof(f,f, &idummy); // Fixme: missing function __nv_remquof
    b[i] += rhypotf(f,f);
    b[i] += rintf(f);
    // b[i] += rnorm3df(f,f,f); // Fixme: missing function __nv_rnorm3df
    // b[i] += rnorm4df(f,f,f,f); // Fixme: missing function __nv_rnorm4df
    // b[i] += rnormf(1, &f); // Fixme: missing function __nv_rnormf
    b[i] += roundf(f);
    b[i] += rsqrtf(f);
    //b[i] += scalblnf(f, 1); // Fixme: missing function __nv_scalbnf
    //b[i] += scalbnf(f, 1);  // Fixme: missing function __nv_scalbnf
    b[i] += signbit(f);
    // sincosf(f, &dummy, &dummy2); // Fixme: missing function __nv_sincosf
    // sincospif(f, &dummy, &dummy2); // Fixme: missing function __nv_sincospif
    b[i] += sinf(f);
    b[i] += sinhf(f);
    b[i] += sinpif(f);
    b[i] += sqrtf(f);
    b[i] += tanf(f);
    b[i] += tanhf(f);
    b[i] += tgammaf(f);
    b[i] += truncf(f);
    b[i] += y0f(f);
    b[i] += y1f(f);
    // b[i] += ynf(1,f); // Fixme: missing function __nv_ynf

   // 1.5 Single Presicion Intrinsics

    b[i] += __cosf(f);
    b[i] += __exp10f(f);
    b[i] += __expf(f);
    //    b[i] += __fadd_rd(f, f); // Fixme: missing function __nv_fadd_rd
    //    b[i] += __fadd_rn(f, f); // Fixme: missing function __nv_fadd_rn
    //    b[i] += __fadd_ru(f, f); // Fixme: missing function __nv_fadd_ru
    //    b[i] += __fadd_rz(f, f); // Fixme: missing function __nv_fadd_rz
    //    b[i] += __fdiv_rd(f, f); // Fixme: missing function __nv_fdiv_rd
    //    b[i] += __fdiv_rn(f, f); // Fixme: missing function __nv_fdiv_rn
    //    b[i] += __fdiv_ru(f, f); // Fixme: missing function __nv_fdiv_ru
    //    b[i] += __fdiv_rz(f, f); // Fixme: missing function __nv_fdiv_rz
    b[i] += __fdividef(f, f);
    // b[i] += __fmaf_rd(f, f, f); // Fixme: missing function __nv_fmaf_rd
    // b[i] += __fmaf_rn(f, f, f); // Fixme: missing function __nv_fmaf_rn
    // b[i] += __fmaf_ru(f, f, f); // Fixme: missing function __nv_fmaf_ru
    // b[i] += __fmaf_rz(f, f, f); // Fixme: missing function __nv_fmaf_rz
    // b[i] += __fmul_rd(f, f); // Fixme: missing function: __nv_fmul_rd
    // b[i] += __fmul_rn(f, f); // Fixme: missing function: __nv_fmul_rn
    // b[i] += __fmul_ru(f, f); // Fixme: missing function: __nv_fmul_ru
    // b[i] += __fmul_rz(f, f); // Fixme: missing function: __nv_fmul_rz
    // b[i] += __frcp_rd(f); // Fixme: missing function: __nv_frcp_rd
    // b[i] += __frcp_rn(f); // Fixme: missing function: __nv_frcp_rn
    // b[i] += __frcp_ru(f); // Fixme: missing function: __nv_frcp_ru
    // b[i] += __frcp_rz(f); // Fixme: missing function: __nv_frcp_rz
    // b[i] += __fsqrt_rd(f); // Fixme: missing function: __nv_fsqrt_rd
    // b[i] += __fsqrt_rn(f); // Fixme: missing function: __nv_fsqrt_rn
    // b[i] += __fsqrt_ru(f); // Fixme: missing function: __nv_fsqrt_ru
    // b[i] += __fsqrt_rz(f); // Fixme: missing function: __nv_fsqrt_rz
    // b[i] += __fsub_rd(f, f); // Fixme: missinf function: __nv_fsub_rd
    b[i] += __log10f(f);
    b[i] += __log2f(f);
    b[i] += __logf(f);
    b[i] += __powf(f, f);
    b[i] += __saturatef(f);
    // __sincosf(f, &dummy, &dummy2); // Fixme: indirect call error to __nv_fast_sincosf
    b[i] += __sinf(f);
    b[i] += __tanf(f);
  }
}

void printArray(float *array)
{
  printf("[");
  bool first = true;
  for (int i = 0; i<N; ++i)
  {
    if (first)
    {
      printf("%f", array[i]);
      first = false;
    }
    else
    {
      printf(", %f", array[i]);
    }
  }
  printf("]");
}

void printHipError(hipError_t error)
{
  printf("Hip Error: %s\n", hipGetErrorString(error));
}

bool hipCallSuccessful(hipError_t error)
{
  if (error != hipSuccess)
    printHipError(error);
  return error == hipSuccess;
}

bool deviceCanCompute(int deviceID)
{
  bool canCompute = false;
  hipDeviceProp_t deviceProp;
  bool devicePropIsAvailable =
    hipCallSuccessful(hipGetDeviceProperties(&deviceProp, deviceID));
  if (devicePropIsAvailable)
  {
    canCompute = deviceProp.computeMode != hipComputeModeProhibited;
    if (!canCompute)
      printf("Compute mode is prohibited\n");
  }
  return canCompute;
}

bool deviceIsAvailable(int *deviceID)
{
  return hipCallSuccessful(hipGetDevice(deviceID));
}

// We always use device 0
bool haveComputeDevice()
{
  int deviceID = 0;
  return deviceIsAvailable(&deviceID) && deviceCanCompute(deviceID);
}

int main()
{

  float hostArray[N];

  if (!haveComputeDevice())
  {
    printf("No compute device available\n");
    return 0;
  }

  for (int i = 0; i<N; ++i)
    hostArray[i] = 0.0;

  printf("Array content before kernel:\n");
  printArray(hostArray);
  printf("\n");

  float *deviceArray;
  if (!hipCallSuccessful(hipMalloc((void **)&deviceArray, N*sizeof(float))))
  {
    printf("Unable to allocate device memory\n");
    return 0;
  }

  hipLaunchKernelGGL((testFloatMath), dim3(N), dim3(1), 0, 0, deviceArray);

  if (hipCallSuccessful(hipMemcpy(hostArray,
                                     deviceArray,
                                     N * sizeof(float),
                                     hipMemcpyDeviceToHost)))
  {
    printf("Array content after kernel:\n");
    printArray(hostArray);
    printf("\n");
  }
  else
  {
    printf("Unable to copy memory from device to host\n");
  }

  hipFree(deviceArray);
  return 0;
}
